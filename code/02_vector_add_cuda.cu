#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"


__global__
void vector_add(const int *vec1, const int *vec2, int *result, const size_t size) {
    int thread_id = theadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id > size)
        return;

    result[thread_id] = vec1[thread_id] + vec2[thread_id];
}


int main(int argc, char *argv[]) {
    size_t num_elements = 10000;
    if (argc > 1) { sscanf(argv[1], "%zu", &num_elements); }
    size_t num_bytes = sizeof(int) * num_elements;

    // ---- DECLARE VARIABLES -----------------------
    int *host_a, *host_b, *host_r;
    int *dev_a, *dev_b, *dev_r;

    // ---- ALLOCATE MEMORY ON HOST -----------------
    host_a = (int *)malloc(num_bytes);
    host_b = (int *)malloc(num_bytes);
    host_r = (int *)malloc(num_bytes);

    // ---- INITIALIZE DATA ON HOST -----------------
    fill_array_constant(host_a, num_elements, 1);
    fill_array_constant(host_b, num_elements, 2);

    // ---- ALLOCATE MEMORY ON DEVICE ---------------
    // hipMalloc(void **dev_ptr, size_t count)
    hipMalloc(&dev_a, num_bytes);
    hipMalloc(&dev_b, num_bytes);
    hipMalloc(&dev_r, num_bytes);

    // ---- COPY DATA OVER TO DEVICE ----------------
    // hipMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
    hipMemcpy(dev_a, host_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, num_bytes, hipMemcpyHostToDevice);

    // ---- PERFORM COMPUTATION ---------------------
    // ... skip over nThreads, nBlocks etc ...
    vector_add<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, dev_r, num_elements);

    // ---- COPY RESULT DATA BACK TO HOST -----------
    hipMemcpy(host_r, dev_r, num_bytes, hipMemcpyDeviceToHost);

    // ---- DO SOMETHING WITH THE DATA --------------
    print_array(host_r, num_elements);

    // ---- CLEANUP ---------------------------------
    free(host_a); free(host_b); free(host_r);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_r);

    return 0;
}

